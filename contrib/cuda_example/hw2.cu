#include <iostream>
#include <cmath>
#include <string>
#include <sys/time.h>

#include <hip/hip_runtime.h>


/* simple timer that stores multiple times 
   and can print them afterwards.
   Does not support nested timing calls. 
*/
class Timer
{
public:
    Timer(): n(0) { }
    void start(std::string label) 
    { 
        if (n < 20) { labels[n] = label; gettimeofday(&times[2*n], NULL); }
        else { std::cerr << "No more timers, " << label << " will not be timed." << std::endl; }
    }

    void stop() { gettimeofday(&times[2*n+1], NULL); n++;}
    void reset() { n=0; }
    void print();
private:
    std::string labels[20];
    timeval times[40];
    int n;
};

void Timer::print()
{
    std::cout << std::endl;
    std::cout << "Action           ::    time/s     Time resolution = " << 1.f/(float)CLOCKS_PER_SEC << std::endl;
    std::cout << "------" << std::endl;
    for (int i=0; i < n; ++i)
    {
        time_t seconds = times[2*i+1].tv_sec - times[2*i+0].tv_sec;
        suseconds_t ms = times[2*i+1].tv_usec - times[2*i+0].tv_usec;
        if (ms < 0) { ms += 1000000; seconds--; }
        std::cout << labels[i] << " ::    " << (float)seconds + ms/1000000.f << std::endl;
    }
}

void initialize( float* x, int n )
{
    int n2 = n+2;
    for (int i=0; i<n2; ++i)
        for (int j=0; j<n2; ++j)
            x[i*n2 + j] = random() / (float) RAND_MAX;
}

void smooth( float* y, float* x, int n, float a, float b, float c )
{
    int n2 = n+2;
    for (int i=1; i<=n; ++i)
        for (int j=1; j<=n; ++j)
            y[i*n2 + j] = a * (x[(i-1)*n2 + (j-1)] + x[(i-1)*n2 + (j+1)] + x[(i+1)*n2 + (j-1)] + x[(i+1)*n2 + (j+1)]) 
                + b * (x[i*n2+(j-1)] + x[i*n2+(j+1)] + x[(i-1)*n2 + j] + x[(i+1)*n2 + j])
                + c * x[i*n2 +j];
}

__global__ void smooth_gpu( float* y, float* x, int n, float a, float b, float c )
{
    int n2 = n+2, i, j;
    
    i = blockDim.x * blockIdx.x + threadIdx.x + 1;  // add one to start from index 1
    j = blockDim.y * blockIdx.y + threadIdx.y + 1;
    
    y[i*n2 + j] = a * (x[(i-1)*n2 + (j-1)] + x[(i-1)*n2 + (j+1)] + x[(i+1)*n2 + (j-1)] + x[(i+1)*n2 + (j+1)]) 
        + b * (x[i*n2+(j-1)] + x[i*n2+(j+1)] + x[(i-1)*n2 + j] + x[(i+1)*n2 + j])
        + c * x[i*n2 +j];
}


void count( float* x, int n, float t, int &nbx )
{
    nbx = 0;
    int n2 = n+2;
    for (int i=1; i <= n; ++i)
        for (int j=1; j <= n; ++j)
            if (x[i*n2 + j] < t) 
                ++nbx;
}


int main( int argc, char** argv )
{
    int n,nbx,nby,nbyh;
    float a,b,c,
        t,diff_y_yh,
        *x, *y, *y_h,
        *x_d, *y_d;
    Timer timer;

    /* n - number of elements in one direction */
    //n = 1 << 14;  /* 2^14 == 16384 */
    n = 1<<12;

    /* a,b,c - smoothing constants */
    a = 0.05;
    b = 0.1;
    c = 0.4;

    /* t - threshold */
    t = 0.1;

    /* allocate x */
    timer.start("CPU: Alloc-X    ");
    x = new float[(n+2)*(n+2)];
    timer.stop();

    /* allocate y and y_h */
    timer.start("CPU: Alloc-Y&Y_H");
    y = new float[(n+2)*(n+2)];
    y_h = new float[(n+2)*(n+2)];
    timer.stop();

    /* allocate device input array x_d */
    timer.start("GPU: Alloc-X_D  ");
    hipMalloc((void**) &x_d, (n+2)*(n+2)*sizeof(float)); 
    timer.stop();

    /* allocate device output array y_d */
    timer.start("GPU: Alloc-Y_D  ");
    hipMalloc((void**) &y_d, (n+2)*(n+2)*sizeof(float)); 
    timer.stop();
    
    /* initialize x */
    timer.start("CPU: Init-X     ");
    initialize(x, n);
    timer.stop();

    /* smooth x into y */
    timer.start("CPU: Smooth     ");
    smooth(y, x, n, a, b, c);
    timer.stop();
    
    /* copy x_h to x_d on GPU */
    timer.start("GPU: Copy-in    ");
    hipMemcpy(x_d, x, (n+2)*(n+2)*sizeof(float), hipMemcpyHostToDevice);
    timer.stop();

    /* launch smooth() on GPU */
    int nBlocks = 16;
    dim3 blockSize(nBlocks, nBlocks);
    dim3 gridSize(n/nBlocks, n/nBlocks);
    timer.start("GPU: Smooth     ");
    smooth_gpu<<<gridSize,blockSize>>>(y_d, x_d, n, a, b, c);
    hipDeviceSynchronize();
    timer.stop();

    /* copy y_d to y_h on CPU */
    timer.start("GPU: Copy-out   ");
    hipMemcpy(y_h, y_d, (n+2)*(n+2)*sizeof(float), hipMemcpyDeviceToHost);
    timer.stop();

    /* count elements in first array */
    timer.start("CPU: Count-X    ");
    count(x, n, t, nbx);
    timer.stop();

    /* count elements in second array */
    timer.start("CPU: Count-Y    ");
    count(y, n, t, nby);
    timer.stop();

    /* count elements in second array from GPU */
    timer.start("CPU: Count-Y_H  ");
    count(y_h, n, t, nbyh);
    timer.stop();

    /* find max abs diff b/t y and y_h */
    diff_y_yh = 0.f;
    int n2 = n+2;
    for (int i=1; i < n; ++i)
        for (int j=1; j < n; ++j)
        {
            float diff = fabsf(y[i*n2 + j] - y_h[i*n2 + j]);
            diff_y_yh = diff > diff_y_yh ? diff : diff_y_yh;
        }

    std::cout << std::endl;
    std::cout << "Summary" << std::endl;
    std::cout << "-------" << std::endl;
    std::cout << "Number of elements in a row/column         :: " << n+2 << std::endl;
    std::cout << "Number of inner elements in a row/column   :: " << n << std::endl;
    std::cout << "Total number of elements                   :: " << (n+2)*(n+2) << std::endl;
    std::cout << "Total number of inner elements             :: " << n*n << std::endl;
    std::cout << "Memory (GB) used per array                 :: " << (n+2)*(n+2)*sizeof(float) / (float)(1024*1024*1024) << std::endl;;
    std::cout << "Threshold                                  :: " << t << std::endl;
    std::cout << "Smoothing constants (a, b, c)              :: " << a << " " << b << " " << c << std::endl;
    std::cout << "Number   of elements below threshold (X)   :: " << nbx << std::endl;
    std::cout << "Fraction of elements below threshold       :: " << nbx / (float)(n*n) << std::endl;
    std::cout << "Number   of elements below threshold (Y)   :: " << nby << std::endl;
    std::cout << "Fraction of elements below threshold       :: " << nby / (float)(n*n) << std::endl;
    std::cout << "Number   of elements below threshold (Y_H) :: " << nbyh << std::endl;
    std::cout << "Fraction of elements below threshold       :: " << nbyh / (float)(n*n) << std::endl;
    std::cout << "Maximum abs. difference between y and y_h  :: " << diff_y_yh << std::endl;
    std::cout << "Block Size                                 :: " << blockSize.x << " " << blockSize.y << " " << blockSize.z << std::endl;
    std::cout << "Grid Size                                  :: " << gridSize.x << " " << gridSize.y << " " << gridSize.z << std::endl;
    std::cout << "Number of threads per block                :: " << blockSize.x * blockSize.y << std::endl;
    std::cout << "Number of blocks per grid                  :: " << gridSize.x * gridSize.y << std::endl;
    std::cout << "Total number of threads                    :: " << blockSize.x * blockSize.y * gridSize.x * gridSize.y << std::endl;

    timer.print();

    delete[] x;
    delete[] y;                                 
    delete[] y_h;
    hipFree(x_d);
    hipFree(y_d);
}
